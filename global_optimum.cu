#include "hip/hip_runtime.h"
#include "global_optimum.h"
#include "limits.h"
#include "stdio.h"
#include "stdlib.h"
int map[RES_COL*RES_ROW];
int *device_map;
size_t pitch;

extern "C" void init(){
  hipMallocPitch(&device_map,&pitch,RES_COL*sizeof(int),RES_ROW);
}

__global__ void calculate_cost(int *device_map ,size_t pitch, int x, int y){
  int thread_x=blockIdx.x*blockDim.x+threadIdx.x;
  int thread_y=blockIdx.y*blockDim.y+threadIdx.y;
  if(thread_x < RES_COL && thread_y< RES_ROW){
    int *cost=(int*)((char*)device_map+thread_y*pitch); 
    cost[thread_x]= (int)(fminf(fabsf(x-thread_x),RES_COL-fabsf(x-thread_x))+fminf(fabsf(y-thread_y),RES_ROW-fabsf(y-thread_y)));
  }
}

extern "C" void update_map(int target_X, int target_y){
  dim3 threadperblock(16,12);
  dim3 blockpergrid(50,50);
  calculate_cost<<<blockpergrid,threadperblock>>>(device_map,pitch ,target_X,target_y);
  hipDeviceSynchronize();
  hipMemcpy2D(map,RES_COL*sizeof(int),device_map,pitch,RES_COL*sizeof(int),RES_ROW,hipMemcpyDeviceToHost);
}
