#include "hip/hip_runtime.h"
#include "Heuristic_Calculation.cuh"
#include "stdio.h"
#include "math.h"
#include "float.h"
const int KERNEL_WIDTH=SEARCH_DEPTH*2+1;
const dim3 TPB(KERNEL_WIDTH,KERNEL_WIDTH,1);
float map[KERNEL_WIDTH*KERNEL_WIDTH];
float *device_map;
size_t pitch;

void init(){
  hipMallocPitch(&device_map,&pitch,KERNEL_WIDTH*sizeof(float),KERNEL_WIDTH);
}

__global__ void d_calculate_distance(float x,float y,int *snake,int length,float *op){
  extern __shared__ float coordinates[];
  int index=threadIdx.x*4;
  float d2,d3;
  coordinates[index]=(float)snake[index];
  coordinates[index+1]=(float)snake[index+1];
  coordinates[index+2]=(float)snake[index+2];
  coordinates[index+3]=(float)snake[index+3];
  d2=((x-coordinates[index])*(coordinates[index+2]-coordinates[index])+(y-coordinates[index+1])*(coordinates[index+3]-coordinates[index+1]))/(powf(coordinates[index]-coordinates[index+2],2)+powf(coordinates[index+1]-coordinates[index+3],2));
  d3=fmaxf(.0f,fminf(1.0f,d2));
  d2=sqrtf(powf(x-(coordinates[index]+d3*(coordinates[index+2]-coordinates[index])),2)+powf(y-(coordinates[index+1]+d3*(coordinates[index+3]-coordinates[index+1])),2));
  if(d2<1.0f){
    *op=-1.0f;
  }
  __syncthreads();
  if(*op==-1.0f){
    return;
  }
  coordinates[length*4]=.0f;
  atomicAdd(&coordinates[length*4],d2);
  __syncthreads();
  if(threadIdx.x==0){
    *op=coordinates[length*4]/(RES_ROW*RES_COL);
  }
}
__global__ void calculate_cost(float *device_map ,size_t pitch, int food_x, int food_y,int *snake,int length,float snake_length){
  int X=GET_X(snake[0],((int)(threadIdx.x-SEARCH_DEPTH)));
  int Y=GET_Y(snake[1],((int)(threadIdx.y-SEARCH_DEPTH)));
  float x1,y1,x2,y2;
  float *cost=(float*)(((char*)device_map)+threadIdx.y*pitch); 
  cost[threadIdx.x]= fminf(fabsf(food_x-X),((float)RES_COL)-fabsf(food_x-X)) + fminf(fabsf(food_y-Y),((float)RES_ROW)-fabsf(food_y-Y));
  float x=(float)X,y=(float)Y;
  /*if(length>8){
    float *opt;
    hipMalloc((void**)&opt,sizeof(float));
    d_calculate_distance<<<1,length,sizeof(float)*(length*4+1)>>>(x,y,snake,length,opt);
    hipDeviceSynchronize();
    cost[threadIdx.x]=*opt;
  }else{
    */
    float  op=0,d2,d3;
    for(int i=0;i<length*4;i+=4){
      x1=(float)snake[i];
      y1=(float)snake[i+1];
      x2=(float)snake[i+2];
      y2=(float)snake[i+3];
      d2=((x-x1)*(x2-x1)+(y-y1)*(y2-y1))/(powf(x1-x2,2)+powf(y1-y2,2));
      d3=fmaxf(.0f,fminf(1.0f,d2));
      d2=sqrtf(powf(x-(x1+d3*(x2-x1)),2)+powf(y-(y1+d3*(y2-y1)),2));
      if(d2<1.0f){
        cost[threadIdx.x]=FLT_MAX;
        return;
      }
      op+=d2;
    }
    cost[threadIdx.x]=cost[threadIdx.x]*(1.0f-op/(RES_COL*RES_ROW));
  //}
}

void update_map(float snake_length){
  RenderingLines *temp=head;
  int *snake_h=NULL,*snake_d=NULL,length=0;
  while(temp!=NULL){
    temp=temp->next;
    length++;
  }
  hipMalloc((void**)&snake_d,sizeof(int)*length*4);
  snake_h=(int*)malloc(sizeof(int)*length*4);
  preprocess(snake_h,length);
  hipMemcpy((void*)snake_d,(void*)snake_h,sizeof(int)*length*4,hipMemcpyHostToDevice);
  calculate_cost<<<1,TPB>>>(device_map,pitch ,f->x1,f->y1,snake_d,length,snake_length);
  hipDeviceSynchronize();
  hipMemcpy2D((void*)map,KERNEL_WIDTH*sizeof(float),(void*)device_map,pitch,KERNEL_WIDTH*sizeof(float),KERNEL_WIDTH,hipMemcpyDeviceToHost);
  /*printf("\n\nIN UPDATE MAP Head at %d %d FOOD at %d %d\n",head->attr->x1,head->attr->y1,f->x1,f->y1);
  printf("\n");
  for(int i=0;i<KERNEL_WIDTH;i++){
    int slice=KERNEL_WIDTH*(KERNEL_WIDTH+i);
    for(int j=0;j<KERNEL_WIDTH;j++){
      printf("%+lf ",(map+slice)[j]);
    }
    printf("\n");
  }
  printf("\n\n");
  */
  hipFree(snake_d);
  free(snake_h);
  snake_h=NULL;

}

void preprocess(int *array,int length){
  RenderingLines *temp=head;
  int i=0;
  while(temp!=NULL){
    array[i]=temp->attr->x1;
    array[i+1]=temp->attr->y1;
    array[i+2]=temp->attr->x2;
    array[i+3]=temp->attr->y2;
    i+=4;
    temp=temp->next;
  }
 /* printf("Points ");
  for(int i=0;i<length*4;i++){
    printf("%d ",array[i]);
    if((i+1)%4==0){
      printf("\n");
    }
  }
  printf("\n");
  */
}