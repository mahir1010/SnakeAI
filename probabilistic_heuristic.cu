#include "hip/hip_runtime.h"
#include "probabilistic_heuristic.h"
#include "float.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#define GET_LEFT(x)(x<=0?RES_COL-1:x-1)
#define GET_RIGHT(x)(x>=RES_COL-1?0:x+1)
#define GET_ABOVE(x)(x<=0?RES_ROW-1:x-1)
#define GET_BELOW(x)(x>=RES_ROW-1?0:x+1)

float map[RES_COL*RES_ROW*2];
float *device_map;
size_t pitch;


extern "C" void init(){
  hipMallocPitch(&device_map,&pitch,RES_COL*sizeof(float),RES_ROW*2);
}

__global__ void calculate_cost(float *device_map ,size_t pitch, int food_x, int food_y,int *snake,int length,float snake_length){
  int thread_x=blockIdx.x*blockDim.x+threadIdx.x;
  int thread_y=blockIdx.y*blockDim.y+threadIdx.y;
  if(thread_x < RES_COL && thread_y< RES_ROW){
    float *cost=(float*)(((char*)device_map)+thread_y*pitch); 
    cost[thread_x]= (fminf(fabsf(food_x-thread_x),RES_COL-fabsf(food_x-thread_x))+fminf(fabsf(food_y-thread_y),RES_ROW-fabsf(food_y-thread_y)));
    cost=(float*)(((char*)device_map)+thread_y*pitch+RES_ROW*pitch);
    float x1,y1,x2,y2;
    float op=0,d1,d2,d3,x=(float)thread_x,y=(float)thread_y;
    for(int i=0;i<length;i+=2){
      x1=(float)snake[i];
      y1=(float)snake[i+1];
      x2=(float)snake[i+2];
      y2=(float)snake[i+3];
      d1=powf(x1-x2,2)+powf(y1-y2,2);
      d2=((x-x1)*(x2-x1)+(y-y1)*(y2-y1))/d1;
      d3=fmaxf(.0f,fminf(1.0f,d2));
      d2=sqrtf(powf(x-(x1+d3*(x2-x1)),2)+powf(y-(y1+d3*(y2-y1)),2));
      if(d2==0){
        cost[thread_x]=-1.0f;
        return;
      }
      op+=d2;
    }
    cost[thread_x]=op/snake_length;
  }
}

extern "C" void update_map(float snake_length){
  dim3 threadperblock(16,12);
  dim3 blockpergrid(50,50);
  RenderingLines *temp=head;
  int *snake_h=NULL,*snake_d=NULL,length=1;
  while(temp!=NULL){
    length++;
    temp=temp->next;
  }
  hipMalloc((void**)&snake_d,sizeof(int)*length*2);
  snake_h=(int*)malloc(sizeof(int)*length*2);
  preprocess(snake_h,length);
  hipMemcpy((void*)snake_d,(void*)snake_h,sizeof(int)*length*2,hipMemcpyHostToDevice);
  printf("snake Length=%lf\n",snake_length);
  calculate_cost<<<blockpergrid,threadperblock>>>(device_map,pitch ,f->x1,f->y1,snake_d,length,snake_length);
  hipDeviceSynchronize();


  hipMemcpy2D(map,RES_COL*sizeof(float),device_map,pitch,RES_COL*sizeof(float),RES_ROW*2,hipMemcpyDeviceToHost);
  
  for(int j=head->attr->y1>10?head->attr->y1-10:0;j<RES_ROW && j<head->attr->y1+10;j++){
    for(int i=head->attr->x1>10?head->attr->x1-10:0;i<RES_COL && i<head->attr->x1+10;i++){
      printf("%+.4lf ",((map+((int)RES_COL*j)+(int)RES_ROW*(int)RES_COL)[i]));  
    }
    printf("\n");
  }
  hipFree(snake_d);
  free(snake_h);
  snake_h=NULL;

}

extern "C" float get_cost(int x,int y){
  return ((map+(RES_COL*y))[x]);
}
extern "C" float get_prob(int x,int y){
  return ((map+(RES_COL*y)+RES_ROW*RES_COL)[x]);
}
extern "C" float calculate_prob(int x,int y,Command c){
  int X,Y;
  float l[5];
  l[0]=get_prob(x,y);
  if(l[0]==-1.0f){
    return 9999.0f;
  }
  X=x<=0?RES_COL-1:x-1;
  l[1]=get_prob(X,y);
  X=x>=RES_COL-1?0:x+1;
  l[2]=get_prob(X,y);
  Y=y<=0?RES_ROW-1:y-1;
  l[3]=get_prob(x,Y);
  Y=y>=RES_ROW-1?0:y+1;
  l[4]=get_prob(x,Y);
  for(int i=1;i<5;i++){
    if(l[i]==-1.0f){
      l[i]=1;
    }
  }
  switch(c){
    case LEFT:
    l[2]=0;
    break;
    case RIGHT:
    l[1]=0;
    break;
    case UP:
    l[4]=0;
    break;
    case DOWN:
    l[3]=0;
    break;
  }
  float sum=0;
  for(int i=0;i<5;i++){
    sum+=l[i];
  }
  sum=sum/4.0f;

  return sum; 
}
extern "C" Command next_command(){
  int x=head->attr->x1;
  int y=head->attr->y1;
  float cost[4],normalizer=0;
  int index=0;
  Command next=LEFT;
  cost[0] = get_cost(GET_LEFT(x), y);
  cost[1] = get_cost(x, GET_ABOVE(y));
  cost[2] = get_cost(GET_RIGHT(x), y);
  cost[3] = get_cost(x, GET_BELOW(y));
  index=0;
  for (int i = 0; i < 4; i++) {
    if(cost[i]<cost[index]){
      index=i;
    }
  }
  normalizer=cost[index];
  for (int i = 0; i < 4; i++) {
    cost[i] = cost[i]/normalizer;
  }

  index=0;
  cost[0] *= calculate_prob(GET_LEFT(x), y, LEFT);
  if (cost[0] < cost[index]) {
    index = 0;
    next = LEFT;
  }
  printf("\nLEFT COST=%lf PROB=%lf MINCOST=%lf",cost[0],get_prob(GET_LEFT(x),y),cost[index]);  

  cost[1] *= calculate_prob(x, GET_ABOVE(y), UP);
  if (cost[1] < cost[index]) {
    index = 1;
    next = UP;
  }
  printf("\nUP COST=%lf PROB=%lf MINCOST=%lf",cost[1],get_prob(x, GET_ABOVE(y)),cost[index]);

  cost[2] *= calculate_prob(GET_RIGHT(x), y, RIGHT);
  if (cost[2] < cost[index]) {
    index = 2;
    next = RIGHT;
  }
  printf("\nRIGHT COST=%lf PROB=%lf MINCOST=%lf",cost[2],get_prob(GET_RIGHT(x), y),cost[index]);

 
  cost[3] *= calculate_prob(x, GET_BELOW(y), DOWN);
  if (cost[3] < cost[index]) {
    index = 3;
    next = DOWN;
  }
  printf("\nDOWN COST=%lf PROB=%lf MINCOST=%lf\n",cost[3],get_prob(x, GET_BELOW(y)),cost[index]);
  return next;
}

extern "C" void preprocess(int *array,int length){
  RenderingLines *temp=head;
  array[0]=temp->attr->x1;
  array[1]=temp->attr->y1;
  array[2]=temp->attr->x2;
  array[3]=temp->attr->y2;
  temp=temp->next;
  int i=2;
  while(temp!=NULL){
    array[i]=temp->attr->x1;
    array[i+1]=temp->attr->y1;
    i+=2;
    temp=temp->next;
  }
  array[(length*2)-2]=last->attr->x2;
  array[(length*2)-1]=last->attr->y2;
  }